#include "hip/hip_runtime.h"
// References: https://github.com/xpn/CUDA-MD5-Crack, https://github.com/iryont/md5-cracker

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#include "assert.cu"
#include "md5.cu"

#define CONST_WORD_LIMIT 10
#define CONST_CHARSET_LIMIT 100
#define CONST_CHARSET "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789"
#define CONST_CHARSET_LENGTH (sizeof(CONST_CHARSET) - 1)

#define CONST_WORD_LENGTH_MIN 1
#define CONST_WORD_LENGTH_MAX 8

#define TOTAL_BLOCKS 16384UL
// #define TOTAL_THREADS 512UL
#define HASHES_PER_KERNEL 32UL

/* Global variables */
uint8_t g_wordLength;

char g_word[CONST_WORD_LIMIT];
char g_charset[CONST_CHARSET_LIMIT];
char g_cracked[CONST_WORD_LIMIT];

__device__ char g_deviceCharset[CONST_CHARSET_LIMIT];
__device__ char g_deviceCracked[CONST_WORD_LIMIT];

__device__ __host__ bool next(uint8_t *length, char *word, uint32_t increment)
{
  uint32_t idx = 0;
  uint32_t add = 0;

  while (increment > 0 && idx < CONST_WORD_LIMIT)
  {
    if (idx >= *length && increment > 0)
    {
      increment--;
    }

    add = increment + word[idx];
    word[idx] = add % CONST_CHARSET_LENGTH;
    increment = add / CONST_CHARSET_LENGTH;
    idx++;
  }

  if (idx > *length)
  {
    *length = idx;
  }

  if (idx > CONST_WORD_LENGTH_MAX)
  {
    return false;
  }

  return true;
}

__global__ void md5Crack(uint8_t wordLength, char *charsetWord, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04)
{
  uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;

  /* Shared variables */
  __shared__ char sharedCharset[CONST_CHARSET_LIMIT];

  /* Thread variables */
  char threadCharsetWord[CONST_WORD_LIMIT];
  char threadTextWord[CONST_WORD_LIMIT];
  uint8_t threadWordLength;
  uint32_t threadHash01, threadHash02, threadHash03, threadHash04;

  /* Copy everything to local memory */
  memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
  memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));
  memcpy(sharedCharset, g_deviceCharset, sizeof(uint8_t) * CONST_CHARSET_LIMIT);

  /* Increment current word by thread index */
  next(&threadWordLength, threadCharsetWord, idx);

  for (uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++)
  {
    for (uint32_t i = 0; i < threadWordLength; i++)
    {
      threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
    }

    md5Hash((unsigned char *)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

    if (threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04)
    {
      memcpy(g_deviceCracked, threadTextWord, threadWordLength);
    }

    if (!next(&threadWordLength, threadCharsetWord, 1))
    {
      break;
    }
  }
}

int main(int argc, char *argv[])
{
  /* Check arguments */
  if (argc != 3 || strlen(argv[1]) != 32)
  {
    std::cout << argv[0] << " <md5_hash> <threads_per_block>" << std::endl;
    return -1;
  }

  const unsigned long TOTAL_THREADS = atoi(argv[2]);

  /* Amount of available devices */
  int devices;
  ERROR_CHECK(hipGetDeviceCount(&devices));

  /* Sync type */
  ERROR_CHECK(hipSetDeviceFlags(hipDeviceScheduleSpin));

  /* Display amount of devices */
  std::cout << "Notice: " << devices << " device(s) found" << std::endl;

  /* Hash stored as u32 integers */
  uint32_t md5Hash[4];

  /* Parse argument */
  for (uint8_t i = 0; i < 4; i++)
  {
    char tmp[16];

    strncpy(tmp, argv[1] + i * 8, 8);
    sscanf(tmp, "%x", &md5Hash[i]);
    md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
  }

  /* Fill memory */
  memset(g_word, 0, CONST_WORD_LIMIT);
  memset(g_cracked, 0, CONST_WORD_LIMIT);
  memcpy(g_charset, CONST_CHARSET, CONST_CHARSET_LENGTH);

  /* Current word length = minimum word length */
  g_wordLength = CONST_WORD_LENGTH_MIN;

  /* Main device */
  hipSetDevice(0);

  /* Time */
  hipEvent_t clockBegin;
  hipEvent_t clockLast;

  hipEventCreate(&clockBegin);
  hipEventCreate(&clockLast);
  hipEventRecord(clockBegin, 0);

  /* Current word is different on each device */
  char **words = new char *[devices];

  for (int device = 0; device < devices; device++)
  {
    hipSetDevice(device);

    /* Copy to each device */
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(uint8_t) * CONST_CHARSET_LIMIT, 0, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyHostToDevice));

    /* Allocate on each device */
    ERROR_CHECK(hipMalloc((void **)&words[device], sizeof(uint8_t) * CONST_WORD_LIMIT));
  }

  while (true)
  {
    bool result = false;
    bool found = false;

    for (int device = 0; device < devices; device++)
    {
      hipSetDevice(device);

      /* Copy current data */
      ERROR_CHECK(hipMemcpy(words[device], g_word, sizeof(uint8_t) * CONST_WORD_LIMIT, hipMemcpyHostToDevice));

      /* Start kernel */
      md5Crack<<<TOTAL_BLOCKS, TOTAL_THREADS>>>(g_wordLength, words[device], md5Hash[0], md5Hash[1], md5Hash[2], md5Hash[3]);

      /* Global increment */
      result = next(&g_wordLength, g_word, TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS);
    }

    /* Display progress */
    char word[CONST_WORD_LIMIT];

    for (int i = 0; i < g_wordLength; i++)
    {
      word[i] = g_charset[g_word[i]];
    }

    std::cout << "Notice: currently at " << std::string(word, g_wordLength) << " (" << (uint32_t)g_wordLength << ")" << std::endl;

    for (int device = 0; device < devices; device++)
    {
      hipSetDevice(device);

      /* Synchronize now */
      hipDeviceSynchronize();

      /* Copy result */
      ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyDeviceToHost));

      /* Check result */
      if (found = *g_cracked != 0)
      {
        std::cout << "Notice: cracked " << g_cracked << std::endl;
        break;
      }
    }

    if (!result || found)
    {
      if (!result && !found)
      {
        std::cout << "Notice: found nothing (host)" << std::endl;
      }

      break;
    }
  }

  for (int device = 0; device < devices; device++)
  {
    hipSetDevice(device);

    /* Free on each device */
    hipFree((void **)words[device]);
  }

  /* Free array */
  delete[] words;

  /* Main device */
  hipSetDevice(0);

  float milliseconds = 0;

  hipEventRecord(clockLast, 0);
  hipEventSynchronize(clockLast);
  hipEventElapsedTime(&milliseconds, clockBegin, clockLast);
  // std::cout << milliseconds << std::endl;

  std::cout << "Notice: computation time " << milliseconds << " ms" << std::endl;

  hipEventDestroy(clockBegin);
  hipEventDestroy(clockLast);
}