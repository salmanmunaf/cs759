#define ERROR_CHECK(X) { gpuAssert((X), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true){
  if(code != hipSuccess){
    std::cout << "Error: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if(abort){
      exit(code);
    }
  }
}