#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <stdio.h>
#include <random>
#include <cmath>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include "cub/util_debug.cuh"
using namespace hipcub;
CachingDeviceAllocator g_allocator(true);  // Caching allocator for device memory

const size_t NUM_PARAM = 2;

int main(int argc, char* argv[]) {
    if (argc != NUM_PARAM) {
        std::cerr << "Invalid parameters" << std::endl;
        return 1;
    }

    const size_t num_items = atoi(argv[1]);

    //Initialize random number generator for array h_in
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source()); 
    std::uniform_real_distribution<float> dist(-1., 1.);

    // Set up host arrays
    float* h_in = new float[num_items];
    for (unsigned int i = 0; i < num_items; i++) {
        h_in[i] = dist(generator);
    }

    // Set up device arrays
    float* d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_in, sizeof(float) * num_items));
    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(float) * num_items, hipMemcpyHostToDevice));
    // Setup device output array
    float* d_sum = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_sum, sizeof(float) * 1));
    // Request and allocate temporary storage
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, num_items));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Do the actual reduce operation
    //Setup timers and record the time it takes for reduce function
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, num_items));
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, num_items);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // Get the elapsed time in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float gpu_sum;
    CubDebugExit(hipMemcpy(&gpu_sum, d_sum, sizeof(float) * 1, hipMemcpyDeviceToHost));
    // Print out results
    std::cout << gpu_sum << std::endl;
    std::cout << milliseconds << std::endl;

    // Cleanup
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_sum) CubDebugExit(g_allocator.DeviceFree(d_sum));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    
    return 0;
}