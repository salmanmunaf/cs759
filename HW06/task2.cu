#include <hip/hip_runtime.h>
#include <random>
#include <iostream>
#include "scan.cuh"

const size_t NUM_PARAM = 3;

int main(int argc, char* argv[]) {
    if (argc != NUM_PARAM) {
        std::cerr << "Invalid parameters" << std::endl;
        return 1;
    }

    const unsigned int N = atoi(argv[1]);
    const unsigned int THREADS_PER_BLOCK = atoi(argv[2]);

    //Generating a random numbers for array a and b
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source()); 
    
    std::uniform_real_distribution<float> dist(-1., 1.);

    // Initialize and allocate device memory for input and output
    float *input, *output;
    hipMallocManaged((void **)&input, N * sizeof(float));
    hipMallocManaged((void **)&output, N * sizeof(float));

    for (unsigned int i = 0; i < N; i++) {
        input[i] = dist(generator);
        output[i] = 0;
    }

    //Setup timers and record the time it takes for reduce function
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    scan(input, output, N, THREADS_PER_BLOCK);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventSynchronize(stop);
    // Get the elapsed time in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    //print out the result to confirm that things are looking good
    std::cout << output[N-1] << std::endl;
    std::cout << milliseconds << std::endl;

    // Cleanup after kernel execution
    hipFree(input);
    hipFree(output);

    return 0;
}