#include "mmul.h"
#include <iostream>

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n) {
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasStatus_t stat;
    //Call cuBLAS library function with appropriate parameters and print message if a call fails
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, B, n, &beta, C, n);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        std::printf("CUBLAS matrix multiplication failed\n");
        return;
    }
}